// -*- C++ -*-

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define CUDA_ERR_CHECK(x) \
    { hipError_t err = x; if (err != hipSuccess) {                \
            fprintf(stderr, "Error \"%s\" at %s:%d \n",	    \
                    hipGetErrorString(err),                        \
                    __FILE__, __LINE__); exit(-1);                  \
        }}

double copyit(double* vertices, int num_vertices)
{

    size_t available, total;
    CUDA_ERR_CHECK(hipMemGetInfo(&available, &total));

    printf("Available: %luMB\nTotal: %luMB\n", available>>20, total>>20);

    double* cuda_vertices;

    double *cpyVerts = (double *)malloc(num_vertices*3*sizeof(double));

   CUDA_ERR_CHECK(hipMalloc(&cuda_vertices, num_vertices*3*sizeof(double)));

    // copy A,B to GPU
    CUDA_ERR_CHECK(hipMemcpy(cuda_vertices, vertices, num_vertices*3*sizeof(double), hipMemcpyHostToDevice));

    CUDA_ERR_CHECK(hipMemcpy(cpyVerts, cuda_vertices, num_vertices*3*sizeof(double), hipMemcpyDeviceToHost));
    
    double *v, *vo = &cpyVerts[0];	
    double ans = 0;    
    for(v = vo; v<vo+num_vertices*3; v+=3)
    {
	printf("%f\n",*(v+1));
	ans += *(v+1);
    }
    return ans;
}
